#include "hip/hip_runtime.h"
/******************************************************************************
** This material was prepared as an account of work sponsored by an agency   **
** of the United States Government.  Neither the United States Government    **
** nor the United States Department of Energy, nor Battelle, nor any of      **
** their employees, nor any jurisdiction or organization that has cooperated **
** in the development of these materials, makes any warranty, express or     **
** implied, or assumes any legal liability or responsibility for the accuracy,* 
** completeness, or usefulness or any information, apparatus, product,       **
** software, or process disclosed, or represents that its use would not      **
** infringe privately owned rights.                                          **
**                                                                           **
** Reference herein to any specific commercial product, process, or service  **
** by trade name, trademark, manufacturer, or otherwise does not necessarily **
** constitute or imply its endorsement, recommendation, or favoring by the   **
** United States Government or any agency thereof, or Battelle Memorial      **
** Institute. The views and opinions of authors expressed herein do not      **
** necessarily state or reflect those of the United States Government or     **
** any agency thereof.                                                       **
**                                                                           **
**                      PACIFIC NORTHWEST NATIONAL LABORATORY                **
**                                  operated by                              **
**                                    BATTELLE                               **
**                                     for the                               **
**                      UNITED STATES DEPARTMENT OF ENERGY                   **
**                         under Contract DE-AC05-76RL01830                  **
**                                                                           **
** Copyright 2019 Battelle Memorial Institute                                **
** Licensed under the Apache License, Version 2.0 (the "License");           **
** you may not use this file except in compliance with the License.          **
** You may obtain a copy of the License at                                   **
**                                                                           **
**    https://www.apache.org/licenses/LICENSE-2.0                            **
**                                                                           **
** Unless required by applicable law or agreed to in writing, software       **
** distributed under the License is distributed on an "AS IS" BASIS, WITHOUT **
** WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the  **
** License for the specific language governing permissions and limitations   **
******************************************************************************/
#include <stdio.h>
#include <stdlib.h>
#include "arts/arts.h"
#include "artsGpuRuntime.h"
#include "mmUtil.h"

#include "hipblas.h"
#include <hip/hip_runtime.h>

#define MATSIZE 1024
#define TILESIZE 32
// #define VERIFY 1
#define SMTILE 32

uint64_t start = 0;

int matSize;
int tileSize;
unsigned int numBlocks = 1;

artsGuid_t aMatGuid = NULL_GUID;
artsGuid_t bMatGuid = NULL_GUID;
artsGuid_t cMatGuid = NULL_GUID;
artsGuid_t doneGuid = NULL_GUID;

double * aMatrix = NULL;
double * bMatrix = NULL;
double * cMatrix = NULL;

artsGuidRange * aTileGuids = NULL;
artsGuidRange * bTileGuids = NULL;

hipblasHandle_t * handle;

void multiplyMM(uint32_t paramc, uint64_t * paramv, uint32_t depc, artsEdtDep_t depv[])
{
    artsGuid_t toSignal = paramv[0];
    unsigned int size = sizeof(double) * tileSize * tileSize;
    // unsigned int i = paramv[1];
    // unsigned int j = paramv[2];
    unsigned int k = paramv[3];
    
    double * aTileDev  = (double*) depv[0].ptr;
    double * bTileDev  = (double*) depv[1].ptr;
    double * cTileHost = NULL;

    // artsGuid_t aTileGuid = depv[0].guid;
    // artsGuid_t bTileGuid = depv[1].guid;
    artsGuid_t cTileGuid = artsDbCreate((void**) &cTileHost, size, ARTS_DB_GPU_WRITE);
    
    double * cTileDev = (double*) artsCudaMalloc(size);

    double alpha  = 1.0;
    double beta = 0.0;

    hipblasDgemm(handle[artsGetGpuId()], HIPBLAS_OP_N, HIPBLAS_OP_N, 
        tileSize, tileSize, tileSize, 
        &alpha, 
        aTileDev, tileSize, 
        bTileDev, tileSize, 
        &beta, 
        cTileDev, tileSize);

    artsPutInDbFromGpu(cTileDev, cTileGuid, 0, size, true);
    artsSignalEdt(toSignal, k, cTileGuid);
}

__global__ void sumMMKernel(uint32_t paramc, uint64_t * paramv, uint32_t depc, artsEdtDep_t depv[])
{
    const unsigned int tileSize = (unsigned int) paramv[0];
    double * cTile = (double *) depv[0].ptr;

    int row = blockDim.x * blockIdx.x + threadIdx.x;
    int col = blockDim.y * blockIdx.y + threadIdx.y;

    for (unsigned int k=1; k<depc; ++k)
    {
        double* toAdd = (double*) depv[k].ptr;
        cTile[row * tileSize + col] += toAdd[row * tileSize + col];
    }
}

void finishBlockMM(uint32_t paramc, uint64_t * paramv, uint32_t depc, artsEdtDep_t depv[])
{
    double * cMat  = (double*) depv[0].ptr;
    for(unsigned int i=0; i<numBlocks; i++)
        for(unsigned int j=0; j<numBlocks; j++)
        {
            double * cTile = (double*) depv[3 + (i * numBlocks + j)].ptr;
            copyBlock(i, j, tileSize, cTile, matSize, cMat, false);
        }

    uint64_t time = artsGetTimeStamp() - start;

#if VERIFY
    double * aMat  = (double*) depv[1].ptr;
    double * bMat  = (double*) depv[2].ptr;
    printf("Verifying results...\n");
    double *temp = (double*) artsCalloc(matSize * matSize * sizeof(double));
    for (unsigned int i=0; i< matSize; ++i)
        for (unsigned int j=0; j<matSize; ++j)
            for (unsigned int k=0; k<matSize; ++k)
                temp[i*matSize+j] += aMat[i*matSize+k]*bMat[k*matSize+j];

    for (unsigned int i=0; i< matSize; ++i)
        for (unsigned int j=0; j<matSize; ++j)
            if (temp[i * matSize + j] != cMat[i * matSize + j])
            {
                printf("Failed at cMat[%u][%u]\n", i, j);
                printf("Expected: %lf | Obtained: %lf\n", temp[i * matSize + j], cMat[i * matSize + j]);
                artsFree(temp);
                artsShutdown();
                return;
            }

    artsFree(temp);
    PRINTF("Success %lu\n", time);
#else
    PRINTF("Done %lu\n", time);
#endif
    artsShutdown();
}

extern "C"
void initPerNode(unsigned int nodeId, int argc, char** argv)
{
    if (argc == 1)
    {
        matSize = MATSIZE;
        tileSize = TILESIZE;
    } else if (argc == 2)
    {
        matSize = atoi(argv[1]);
        tileSize = TILESIZE;
    } else
    {
        matSize = atoi(argv[1]);
        tileSize = atoi(argv[2]);
    }

    numBlocks = matSize / tileSize;
    doneGuid = artsReserveGuidRoute(ARTS_EDT,     0);
    aMatGuid = artsReserveGuidRoute(ARTS_DB_READ, 0);
    bMatGuid = artsReserveGuidRoute(ARTS_DB_READ, 0);
    cMatGuid = artsReserveGuidRoute(ARTS_DB_READ, 0);
    
    aTileGuids = artsNewGuidRangeNode(ARTS_DB_GPU_READ, numBlocks*numBlocks, 0);
    bTileGuids = artsNewGuidRangeNode(ARTS_DB_GPU_READ, numBlocks*numBlocks, 0);
    
    if(!nodeId)
    {
        aMatrix = (double*) artsDbCreateWithGuid(aMatGuid, matSize * matSize * sizeof(double));
        bMatrix = (double*) artsDbCreateWithGuid(bMatGuid, matSize * matSize * sizeof(double));
        cMatrix = (double*) artsDbCreateWithGuid(cMatGuid, matSize * matSize * sizeof(double));
        
        initMatrix(matSize, aMatrix, true, false);
        initMatrix(matSize, bMatrix, false, false);
        initMatrix(matSize, cMatrix, false, true);
        
        PRINTF("Starting\n");
    }
}

extern "C"
void initPerWorker(unsigned int nodeId, unsigned int workerId, int argc, char** argv)
{
    unsigned int totalThreads = artsGetTotalNodes() * artsGetTotalWorkers();
    unsigned int globalThreadId = nodeId * artsGetTotalWorkers() + workerId;   
  
    if(!nodeId && !workerId)
    {
        for(unsigned int i=0; i<numBlocks; i++)
        {
            for(unsigned int j=0; j<numBlocks; j++)
            {
                artsGuid_t aTileGuid = artsGetGuid(aTileGuids, i * numBlocks + j);
                double * aTile = (double*) artsDbCreateWithGuid(aTileGuid, sizeof(double) * tileSize * tileSize);
                copyBlock(i, j, tileSize, aTile, matSize, aMatrix, true);

                artsGuid_t bTileGuid = artsGetGuid(bTileGuids, i * numBlocks + j);
                double * bTile = (double*) artsDbCreateWithGuid(bTileGuid, sizeof(double) * tileSize * tileSize);
                copyBlock(i, j, tileSize, bTile, matSize, bMatrix, true);
            }
        }
    }

    uint64_t sumArgs[] = {tileSize};
    dim3 threads(SMTILE, SMTILE);
    dim3 grid((tileSize+SMTILE-1)/SMTILE, (tileSize+SMTILE-1)/SMTILE);

    for(unsigned int i=0; i<numBlocks; i++)
    {
        for(unsigned int j=0; j<numBlocks; j++)
        {
            if((i * numBlocks + j) % totalThreads == globalThreadId)
            {
                artsGuid_t sumGuid = artsEdtCreateGpuPT (sumMMKernel, nodeId, 1, sumArgs, numBlocks, grid, threads, doneGuid, 3 + (i * numBlocks + j), 0);
                for(unsigned int k=0; k<numBlocks; k++)
                {
                    uint64_t args[] = {sumGuid, i, j, k};
                    artsGuid_t mulGuid = artsEdtCreateGpuLib(multiplyMM, nodeId, 4, args, 2, grid, threads);
                    artsSignalEdt(mulGuid, 0, artsGetGuid(aTileGuids, i * numBlocks + k));
                    artsSignalEdt(mulGuid, 1, artsGetGuid(bTileGuids, k * numBlocks + j));
                }
            }
        }
    }

    if(!nodeId && !workerId)
    {
        artsEdtCreateWithGuid(finishBlockMM, doneGuid, 0, NULL, 3 + numBlocks * numBlocks);
        artsSignalEdt(doneGuid, 0, cMatGuid);
        artsSignalEdt(doneGuid, 1, aMatGuid);
        artsSignalEdt(doneGuid, 2, bMatGuid);
        start = artsGetTimeStamp();
    }
}

extern "C"
void initPerGpu(unsigned int nodeId, int devId, hipStream_t * stream, int argc, char * argv)
{
    if(!devId)
        handle = (hipblasHandle_t*) artsCalloc(sizeof(hipblasHandle_t) * artsGetNumGpus());
    hipblasStatus_t stat = hipblasCreate(&handle[devId]);
}

extern "C"
void cleanPerGpu(unsigned int nodeId, int devId, hipStream_t * stream)
{
    hipblasStatus_t stat = hipblasDestroy(handle[devId]);
}

int main(int argc, char** argv)
{
    artsRT(argc, argv);
    return 0;
}